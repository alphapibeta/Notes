#include "hip/hip_runtime.h"
#include "HeatEquationSolver.h"
#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>

__global__ void gpu_kernel(float *grid, float *new_grid, int N, float alpha, float dt, float dx2, float dy2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < N - 1 && j > 0 && j < N - 1) {
        int index = i * N + j;
        float temp = grid[index];
        float temp_left = grid[i * N + (j - 1)];
        float temp_right = grid[i * N + (j + 1)];
        float temp_top = grid[(i - 1) * N + j];
        float temp_bottom = grid[(i + 1) * N + j];
        new_grid[index] = temp + alpha * dt * ((temp_left - 2.0f * temp + temp_right) / dx2 +
                                               (temp_top - 2.0f * temp + temp_bottom) / dy2);
    } else {
        int index = i * N + j;
        new_grid[index] = grid[index];
    }
}

void gpu_heat_equation_solver(int N, int num_steps, float *grid, float alpha, float dt, float dx2, float dy2, int block_thread_x, int block_thread_y) {
    float *d_grid, *d_new_grid;
    hipError_t err;

    err = hipMalloc(&d_grid, N * N * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA error (d_grid allocation): " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_new_grid, N * N * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA error (d_new_grid allocation): " << hipGetErrorString(err) << std::endl;
        hipFree(d_grid);
        return;
    }

    err = hipMemcpy(d_grid, grid, N * N * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA error (memcpy to device): " << hipGetErrorString(err) << std::endl;
        hipFree(d_grid);
        hipFree(d_new_grid);
        return;
    }

    dim3 threads_per_block(block_thread_x, block_thread_y);
    dim3 num_blocks((N + block_thread_x - 1) / block_thread_x, (N + block_thread_y - 1) / block_thread_y);

    std::cout << "Grid dimensions: " << num_blocks.x << "x" << num_blocks.y << std::endl;
    std::cout << "Block dimensions: " << threads_per_block.x << "x" << threads_per_block.y << std::endl;

    nvtxRangeId_t gpu_solver_range = nvtxRangeStartA("GPU Solver");  
    for (int step = 0; step < num_steps; ++step) {
        gpu_kernel<<<num_blocks, threads_per_block>>>(d_grid, d_new_grid, N, alpha, dt, dx2, dy2);
        
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA error (kernel launch): " << hipGetErrorString(err) << std::endl;
            break;
        }

        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            std::cerr << "CUDA error (device sync): " << hipGetErrorString(err) << std::endl;
            break;
        }

        std::swap(d_grid, d_new_grid);
    }
    nvtxRangeEnd(gpu_solver_range);

    err = hipMemcpy(grid, d_grid, N * N * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA error (memcpy to host): " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_grid);
    hipFree(d_new_grid);
}