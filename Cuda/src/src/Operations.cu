#include "hip/hip_runtime.h"

#include "Operations.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAddKernel(const float *a, const float *b, float *c, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        c[i] = a[i] + b[i];
    }
}

bool cudaOperation(const float *a, const float *b, float *c, int numElements, float &elapsedTime) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    hipMalloc((void **)&d_a, numElements * sizeof(float));
    hipMalloc((void **)&d_b, numElements * sizeof(float));
    hipMalloc((void **)&d_c, numElements * sizeof(float));

    hipMemcpy(d_a, a, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, numElements * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    hipEventRecord(start);
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, numElements);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Check for any errors launching the kernel
    hipError_t error = hipPeekAtLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return false;
    }

    // Copy the device result vector in device memory to the host result vector
    hipMemcpy(c, d_c, numElements * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory and events
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return true;
}




__global__ void runningSumKernel(const float *input, float *output, int numElements) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0.0f;
    for (int j = 0; j <= i && j < numElements; j++) {
        sum += input[j];
    }

    if (i < numElements) {
        output[i] = sum;
    }
}

bool cudaRunningSum(const float *input, float *output, int numElements, float &elapsedTime) {
    float *d_input = nullptr, *d_output = nullptr;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **)&d_input, numElements * sizeof(float));
    hipMalloc((void **)&d_output, numElements * sizeof(float));

    hipMemcpy(d_input, input, numElements * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start);
    runningSumKernel<<<(numElements + 255) / 256, 256>>>(d_input, d_output, numElements);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(output, d_output, numElements * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return true;
}